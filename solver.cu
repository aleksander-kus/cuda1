
#include "hip/hip_runtime.h"


#include <iostream>
#include <fstream>
#include <stdio.h>
#include <string>
#include <chrono>

#define ERR(source) fprintf(stderr, source); goto Error

#define BOARDSIZE 9
#define BLANK 0

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void reduce0(int* g_idata, int* g_odata) {
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void readSudokuFromFile(std::string filepath, int* board)
{
    std::ifstream fileStream(filepath);
    std::string input;
    int index = 0;
    while(getline(fileStream, input))
    {
        for(auto c : input)
        {
            board[index++] = c - '0';
        }
    }
}

void printSudoku(int* board)
{
    const std::string lineBreak = "+-------+-------+-------+\n";
    const std::string columnBreak = "| ";

    for(auto i = 0; i < BOARDSIZE; ++i)
    {
        if(i % 3 == 0)
        {
            std::cout << lineBreak;
        }
        for(auto j = 0; j < BOARDSIZE; ++j)
        {
            if(j % 3 == 0)
            {
                std::cout << columnBreak;
            }

            auto value = board[i * BOARDSIZE + j];
            if(value == BLANK)
            {
                std::cout << ". ";
            }
            else
            {
                std::cout << value << ' ';
            }
        }
        std::cout << columnBreak << std::endl;
    }
    std::cout << lineBreak;
}

bool findEmpty(int* board, int& i, int& j)
{
    for(int k = 0; k < BOARDSIZE; ++k)
    {
        for(int l = 0; l < BOARDSIZE; ++l)
        {
            if(board[k * BOARDSIZE + l] == 0)
            {
                i = k;
                j = l;
                return true;
            }
        }
    }
    return false;
}

bool checkIfCorrectRow(int* board, const int& i, const int& value)
{
    for(int j = 0; j < BOARDSIZE; ++j)
    {
        if(board[i * BOARDSIZE + j] == value)
        {
            return false;
        }
    }
    return true;
}

bool checkIfCorrectColumn(int* board, const int& j, const int& value)
{
    for(int i = 0; i < BOARDSIZE; ++i)
    {
        if(board[i * BOARDSIZE + j] == value)
        {
            return false;
        }
    }
    return true;
}

bool checkIfCorrectBox(int* board, const int& i, const int& j, const int& value)
{
    int rowCenter = (i / 3) * 3 + 1;
    int columnCenter = (j / 3) * 3 + 1;

    for(int k = -1; k < 2; ++k)
    {
        for(int l = -1; l < 2; ++l)
        {
            if(board[(rowCenter + k) * BOARDSIZE + (columnCenter + l)] == value)
            {
                return false;
            }
        }
    }
    return true;
}

bool checkIfCorrect(int* board, int i, int j, int value)
{
    return checkIfCorrectRow(board, i, value) && checkIfCorrectColumn(board, j, value) && checkIfCorrectBox(board, i, j, value);
}

bool solveBacktracking(int* board)
{
    int i = 0;
    int j = 0;

    if(!findEmpty(board, i, j))
    {
        return true;
    }

    for(int x = 1; x < 10; ++x)
    {
        if(checkIfCorrect(board, i, j, x))
        {
            board[i * BOARDSIZE + j] = x;
            if(solveBacktracking(board))
            {
                return true;
            }
            board[i * BOARDSIZE + j] = BLANK;
        }
    }
    return false;
}

int main()
{
    // const int arraySize = 6;
    // const int a[arraySize] = { 1, 2, 3, 4, 5, 6 };
    // //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    // int c[arraySize] = { 0 };

    // Add vectors in parallel.
    //cudaError_t cudaStatus = ReduceWithCuda(c, a, arraySize);
    // if (cudaStatus != cudaSuccess) {
    //     fprintf(stderr, "addWithCuda failed!");
    //     return 1;
    // }

    // printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //     c[0], c[1], c[2], c[3], c[4]);

    // // cudaDeviceReset must be called before exiting in order for profiling and
    // // tracing tools such as Nsight and Visual Profiler to show complete traces.
    // cudaStatus = cudaDeviceReset();
    // if (cudaStatus != cudaSuccess) {
    //     fprintf(stderr, "cudaDeviceReset failed!");
    //     return 1;
    // }
    int board[BOARDSIZE*BOARDSIZE];

    readSudokuFromFile("hard.in", board);
    printSudoku(board);
    std::cout << "Solving sudoku..." << std::endl;
    auto start = std::chrono::high_resolution_clock::now();
    auto result = solveBacktracking(board);
    auto stop = std::chrono::high_resolution_clock::now();
    if(result)
    {
        std::cout << "Sudoku solved!" << std::endl;
        printSudoku(board);
    }
    else
    {
        std::cout << "Could not solve sudoku :(" << std::endl;
    }

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Total time for solving sudoku: " << duration.count() << " microseconds" << std::endl;
    return 0;
}

hipError_t ReduceWithCuda(int* out, const int* a, unsigned int size)
{
    int* dev_input = 0;
    int* dev_output = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);
    if (hipSetDevice(0) != hipSuccess) ERR("hipSetDevice");

    // Allocate GPU buffers for three vectors (two input, one output)    .
    if (hipMalloc((void**)&dev_output, size * sizeof(int)) != hipSuccess) ERR("hipMalloc");    
    if (hipMalloc((void**)&dev_input, size * sizeof(int)) != hipSuccess) ERR("hipMalloc");

    // Copy input vectors from host memory to GPU buffers.
    if (hipMemcpy(dev_input, a, size * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) ERR("hipMemcpy");

    reduce0<<<1, size>>>(dev_input, dev_output);

    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(hipGetLastError()));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
    if (hipDeviceSynchronize() != hipSuccess) ERR("hipDeviceSynchronize");


Error:
    hipFree(dev_input);
    hipFree(dev_output);

    return hipErrorAlreadyAcquired;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
