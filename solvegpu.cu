#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <iostream>
#include <chrono>

#include "solvegpu.cuh"

#define MEMORY_USED 0.01

#define ERR(status) { \
    if (status != hipSuccess) { \
        printf("Error: %s, file: %s, line: %d\n", hipGetErrorString(status), __FILE__,__LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

enum GENERATE_STATUS {
    OK = 0,
    SOLVED = 1,
    OUT_OF_MEMORY = 2
};

__host__ __device__ bool findEmpty(const char* board, int& i, int& j)
{
    for(int k = 0; k < BOARDSIZE; ++k)
    {
        for(int l = 0; l < BOARDSIZE; ++l)
        {
            if(board[k * BOARDSIZE + l] == 0)
            {
                i = k;
                j = l;
                return true;
            }
        }
    }
    return false;
}

__host__ __device__ bool tryToInsertRow(const char* board, const int& i, const char& value)
{
    for(int j = 0; j < BOARDSIZE; ++j)
    {
        if(board[i * BOARDSIZE + j] == value)
        {
            return false;
        }
    }
    return true;
}

__host__ __device__ bool tryToInsertColumn(const char* board, const int& j, const char& value)
{
    for(int i = 0; i < BOARDSIZE; ++i)
    {
        if(board[i * BOARDSIZE + j] == value)
        {
            return false;
        }
    }
    return true;
}

__host__ __device__ bool tryToInsertBox(const char* board, const int& i, const int& j, const char& value)
{
    int rowCenter = (i / 3) * 3 + 1;
    int columnCenter = (j / 3) * 3 + 1;

    for(int k = -1; k < 2; ++k)
    {
        for(int l = -1; l < 2; ++l)
        {
            if(board[(rowCenter + k) * BOARDSIZE + (columnCenter + l)] == value)
            {
                return false;
            }
        }
    }
    return true;
}

__host__ __device__ bool tryToInsert(const char* board, const int& i, const int& j, const char& value)
{
    return value > 0 && value < 10 && tryToInsertRow(board, i, value) && tryToInsertColumn(board, j, value) && tryToInsertBox(board, i, j, value);
}

__device__ void copyBoardToOutput(const char* board, char* output)
{
    for(int i = 0; i < BOARDSIZE; ++i)
    {
        for(int j = 0; j < BOARDSIZE; ++j)
        {
            output[i * BOARDSIZE + j] = board[i * BOARDSIZE + j];
        }
    }
}

__global__ void generate(char* input, char* output, int inputSize, int* outputIndex, int maxOutputSize, GENERATE_STATUS* status)
{
    auto id = blockDim.x * blockIdx.x + threadIdx.x;

    while(id < inputSize && *status == OK)
    {
        int i = 0, j = 0;

        auto board = input + id * BOARDLENGTH; // set the correct input board according to threadIdx
        if(!findEmpty(board, i, j))
        {
            *status = SOLVED;
            return;
        }
        // generate a separate board for all numbers available in the empty spot
        for(int num = 1; num < 10; ++num)
        {
            if(*outputIndex >= maxOutputSize - 1)
            {
                *status = OUT_OF_MEMORY;
                return;
            }
            if(tryToInsert(board, i, j, num))
            {
                board[i * BOARDSIZE + j] = num;
                copyBoardToOutput(board, output + atomicAdd(outputIndex, 1) * BOARDLENGTH);
                board[i * BOARDSIZE + j] = BLANK;
            }
        }
        id += gridDim.x * blockDim.x;
    }
}

__device__ void getEmptyIndices(const char* board, char* indices, char* size)
{
    for(char i = 0; i < BOARDLENGTH; ++i)
    {
        if(board[i] == BLANK)
        {
            indices[*size] = i;
            ++(*size);
        }
    }
}

__global__ void backtrack(char* input, char* output, int inputSize, bool* isSolved)
{
    auto id = blockDim.x * blockIdx.x + threadIdx.x;

    char emptyIndices[BOARDLENGTH];
    char emptyIndicesSize = 0;
    int i = 0, j = 0;

    while(id < inputSize && !*isSolved)
    {
        auto board = input + id * BOARDLENGTH;
        emptyIndicesSize = 0;
        getEmptyIndices(board, emptyIndices, &emptyIndicesSize);
        int index = 0;
        while(index >= 0 && index < emptyIndicesSize)
        {
            auto emptyIndex = emptyIndices[index];
            i = emptyIndex / BOARDSIZE;
            j = emptyIndex % BOARDSIZE;
            // #if __CUDA_ARCH__>=200
            //     printf("Scanning index %d, i = %d, j = %d, value %d \n", emptyIndex, i, j, board[emptyIndex] + 1);
            // #endif
            if(!tryToInsert(board, i, j, board[emptyIndex] + 1))
            {
                if(board[emptyIndex] >= 8)
                {
                    board[emptyIndex] = -1;
                    --index;
                }
            }
            else
            {
                ++index;
            }
            ++board[emptyIndex];
        }

        if(index == emptyIndicesSize)
        {
            *isSolved = true;
            #if __CUDA_ARCH__>=200
                printf("Found solution. index = %d, emptyI = %d \n", index, emptyIndicesSize);
            #endif
            copyBoardToOutput(board, output);
        }
        id += gridDim.x * blockDim.x;
    }
}

int getMaxBoardNumber()
{
    size_t free_memory;
	hipMemGetInfo(&free_memory, nullptr);
    return free_memory * MEMORY_USED / (sizeof(char) * BOARDLENGTH * 2);
}

char* solveGpu(const char* board)
{
    int maxBoardNumber = getMaxBoardNumber();
    char *dev_input = 0, *dev_output = 0;
    int* dev_outputIndex = 0;
    GENERATE_STATUS* dev_status;
    GENERATE_STATUS status;
    int inputSize = 1;
    int oldInputSize = 1;
    int generation = 0;
    int blocks = 2048;
    int threads = 1024;

    ERR(hipMalloc(&dev_input, sizeof(char) * BOARDLENGTH * maxBoardNumber));
    ERR(hipMalloc(&dev_output, sizeof(char) * BOARDLENGTH * maxBoardNumber));
    ERR(hipMalloc(&dev_outputIndex, sizeof(int)));
    ERR(hipMalloc(&dev_status, sizeof(int)));
    ERR(hipMemcpy(dev_input, board, sizeof(char) * BOARDLENGTH, hipMemcpyKind::hipMemcpyHostToDevice));
    ERR(hipMemset(dev_output, 0, sizeof(char) * BOARDLENGTH * maxBoardNumber));

    auto start = std::chrono::high_resolution_clock::now();
    while(generation < 81)
    {
        ERR(hipMemset(dev_outputIndex, 0, sizeof(int)));
        if(generation % 2 == 0)
        {
            generate<<<blocks, threads>>>(dev_input, dev_output, inputSize, dev_outputIndex, maxBoardNumber, dev_status);
        }
        else
        {
            generate<<<blocks, threads>>>(dev_output, dev_input, inputSize, dev_outputIndex, maxBoardNumber, dev_status);
        }
        oldInputSize = inputSize;
        ERR(hipMemcpy(&inputSize, dev_outputIndex, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost));
        ERR(hipMemcpy(&status, dev_status, sizeof(GENERATE_STATUS), hipMemcpyKind::hipMemcpyDeviceToHost));
        if(status != OK || inputSize == 0)
            break;
        ++generation;
    }
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Total time for generating boards: " << duration.count() << " microseconds" << std::endl;
    std::cout << "Finished generating boards with " << inputSize << " boards on generation " << generation << std::endl;

    char* ret = nullptr;
    if (status == SOLVED)
    {
        std::cout << "Sudoku solved by BFS!" << std::endl;
        auto result = generation % 2 == 0 ? dev_input : dev_output; // take the output as result
        ret = (char*)malloc(sizeof(char) * BOARDLENGTH);
        ERR(hipMemcpy(ret, result, sizeof(char) * BOARDLENGTH, hipMemcpyKind::hipMemcpyDeviceToHost));
    }
    else if (inputSize == 0)
    {
        std::cout << "No valid solutions were found for sudoku" << std::endl;
    }
    else
    {
        std::cout << "Available memory exceeded, falling back to last generation of boards and using backtracking" << std::endl;
        auto result = generation % 2 == 1 ? dev_input : dev_output; // take the last input as result
        bool* dev_isSolved;
        char* dev_output_backtracking;
        ERR(hipMalloc(&dev_output_backtracking, sizeof(char) * BOARDLENGTH));

        ERR(hipMalloc(&dev_isSolved, sizeof(bool)));
        ret = (char*)malloc(sizeof(char) * BOARDLENGTH);
        start = std::chrono::high_resolution_clock::now();
        backtrack<<<blocks, threads>>>(result, dev_output_backtracking, oldInputSize, dev_isSolved);
        ERR(hipMemcpy(ret, dev_output_backtracking, sizeof(char) * BOARDLENGTH, hipMemcpyKind::hipMemcpyDeviceToHost));
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        std::cout << "Backtracking took: " << duration.count() << " microseconds" << std::endl;
        ERR(hipFree(dev_isSolved));
    }

    ERR(hipFree(dev_input));
    ERR(hipFree(dev_output));
    ERR(hipFree(dev_outputIndex));
    ERR(hipFree(dev_status));

    return ret;
}